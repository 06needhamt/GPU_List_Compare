#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust\random.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#define SIZE 500

thrust::host_vector<int>* vec = new thrust::host_vector<int>(0);
thrust::device_vector<int>* d_vec = new thrust::device_vector<int>(0);

struct RandomNumberGenerator{
	float a, b;
	RandomNumberGenerator(float _a = 0.0f, float _b = 100.0f) : a(_a), b(_b) {};
	const float operator() (const unsigned int n) const
	{
		thrust::default_random_engine rng;
		thrust::uniform_real_distribution<float> dist(0, 500);
		rng.discard(n);

		return dist(rng);
	}
};

void createVector(){
	for (int i = 0; i < SIZE; i++)
	vec->push_back(RandomNumberGenerator(1.0f, 500.0f)(0));
}
hipError_t copyVector(){
	return hipMemcpy(d_vec, vec, SIZE * sizeof(int), hipMemcpyHostToDevice);
}

__device__ __host__ void compareNumbers(int max){
	for (int i = 0; i < SIZE; i++){
		if (vec[i] < 250){
			printf("%i", d_vec[i]);
		}
	}
}
