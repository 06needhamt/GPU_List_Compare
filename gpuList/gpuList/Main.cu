#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

extern hipError_t copyVector(void);
extern void createVector(void);
extern __global__ void compareNumbers(int max);

int main(int argc, char* argv[]){
	createVector();
	if(copyVector() != hipError_t::hipSuccess){
		std::cout << "An Error Occurred While Copying the vector" << std::endl;
		system("PAUSE");
		return 0;
	}
	compareNumbers << < 1, 500 >> >(500);
	return 0;
}
